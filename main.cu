

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
  printf("Hello, CUDA!\n");
}

int main() {
  hello<<<1, 1>>>();
  hipDeviceSynchronize();
}
